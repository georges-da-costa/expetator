
#include<iostream>
#include <locale>
#include <chrono>
#include "hip/hip_runtime.h"

#define OK(ans) { ok_assert((ans), __FILE__, __LINE__); }
inline void ok_assert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
	   std::cerr<<"error: "<< hipGetErrorString(code)<<", "<< file<<":"<< line <<'\n';
      if (abort) { std::exit(code); }
   }
}
 
struct SEP1K : std::numpunct<char> {
	char do_thousands_sep()   const { return ','; } 
	std::string do_grouping() const { return "\3"; }
};

hipDeviceProp_t device_prop(int devid, int verbose) {
	hipDeviceProp_t prop;
	int maxdev {0};
	OK( hipGetDeviceCount(&maxdev) );
	if (devid >= maxdev) {
		std::cerr<<"fatal: invalid device (gpu) id selected.\n";
		std::exit(1);
	}
	OK( hipSetDevice(devid) );
	OK( hipGetDeviceProperties(&prop, devid) );
	if (verbose) {
		auto prev = std::cout.imbue(std::locale(std::cout.getloc(), new SEP1K));
		std::cout<<prop.name<<" [gpu: "<<devid<<"/"<<maxdev<<"], global mem="<<prop.totalGlobalMem<<'\n';
		std::cout.imbue(prev);
	}
	return prop;
}
void membench(float ratio, size_t sz) {
	// cudamalloc does not oversubscribe as for unified memory.
	size_t sz2 = ratio > .98 ? .98*sz/ratio : sz;
	void *h, *d;
	h = malloc(sz2);
	if (!h) std::exit(2);
	OK( hipMalloc(&d, sz2) );
	OK( hipMemcpy(d, h, sz2, hipMemcpyHostToDevice) );
	OK( hipMemcpy(h, d, sz2, hipMemcpyDeviceToHost) );
}

__device__
unsigned int rand (unsigned int *rand_next) {
       *rand_next = *rand_next * 1103515245 + 12345; 
       return ((unsigned int)(*rand_next / 65536) % 32768);
}

__global__ void busy(int jmax, size_t n, float *data) {
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = gridDim.x * blockDim.x;
	for (size_t i=tid; i<n; i+=stride) {
		float x{0.};
		unsigned int rand_next=i;
		for (unsigned int j=0; j<jmax; ++j) {
			// idea is to have int and float operations (64 cores for each on 7.x)
			x += normcdfinvf(rand(&rand_next));
		}
		data[i] = x;
	}
}
void cpubench(int loop, size_t sz, int blk, int tpb, int verbose) {
	float *data;  // float: 7.x => 64 FP32 cores, 16 special cores.
	size_t n = sz/sizeof(*data);
	OK( hipMallocManaged(&data, n*sizeof(*data)) );
	if (verbose) { 
		auto prev = std::cout.imbue(std::locale(std::cout.getloc(), new SEP1K));
		std::cout<<"mem="<<n*sizeof(*data)<<" B ["<<n<<" x "<<sizeof(*data)<<" B]\n";
		std::cout.imbue(prev);
	}
	busy<<<blk, tpb>>>(loop, n, data);
	OK( hipPeekAtLastError() );
	OK( hipDeviceSynchronize() );
	OK( hipFree(data) );
}

int main(int argc, char *argv[]) {
	int verbose {1}; // default is verbose, -s (silent) removes output
	int rc {EXIT_SUCCESS};
	int bench {0}; // bench type
	float memratio {0.};
	size_t memsz {0};  // memory size
	int tpb {0}; // threads/block
	int blk {0}; // blocks
	int loop{0};
	int ia {1};
	int dev {0}; // target device id 
	if (argc >= 2) { 
		if (std::string(argv[ia]) == "-h")  {
			std::cout<<"arguments: bench [-s] device_id bench_id {0*: mem, 1: cpu} mem_ratio threads/block blocks loop\n"; 
			return EXIT_SUCCESS;
		} else if (std::string(argv[ia]) == "-s") {
			verbose = 0;
			++ia;
		}
	}
	if (ia < argc) { dev = std::stoi(argv[ia++]); }
	if (ia < argc) { bench = std::stoi(argv[ia++]); }
	if (ia < argc) { memratio = std::stof(argv[ia++]); }
	if (ia < argc) { tpb = std::stoi(argv[ia++]); }
	if (ia < argc) { blk = std::stoi(argv[ia++]); }
	if (ia < argc) { loop = std::stoi(argv[ia++]); }
	auto prop = device_prop(dev, verbose);
	if (memratio <= 0.) memratio = .25;
	memsz = prop.totalGlobalMem * memratio;
	if (tpb > prop.maxThreadsPerBlock || tpb <= 0) tpb = prop.maxThreadsPerBlock;
	if (blk <= 0) blk = 2*prop.multiProcessorCount;
	if (loop == 0) loop = 1000;
	if (verbose) {
		std::cout<<"bench "<<bench<<", mem="<<100.0*memratio<<"%, threads/block="<<tpb<<", blocks="<<blk<<'\n';
	}
	auto t0 = std::chrono::high_resolution_clock::now();
	if (bench == 0) membench(memratio, memsz);
	else cpubench(loop, memsz, blk, tpb, verbose); 
	auto t1 = std::chrono::high_resolution_clock::now();
        std::cout<< std::chrono::duration_cast<std::chrono::seconds>(t1-t0).count()<<'\n';

	return rc;
}
